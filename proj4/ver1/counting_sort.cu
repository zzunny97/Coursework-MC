#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void CountSort(int*, int*, int, int);

__host__ void counting_sort(int* arr, int size, int max_val)
{
	int block_num = 4;
	int thread_num_per_block = 1000;
	uint64_t histo_size = sizeof(int)*max_val*block_num;
	printf("size: %d\n", size);
	printf("max_val: %d\n", max_val);
	printf("block_num: %d\n", block_num);
	printf("thread_per_block: %d\n", thread_num_per_block);

	int* dhisto;
	hipMalloc(&dhisto, histo_size);
	hipMemset(dhisto, 0, histo_size);

	int* darr;
	hipMalloc(&darr, sizeof(int)*size);
	hipMemcpy(darr, arr, sizeof(int)*size, hipMemcpyHostToDevice); 

	printf("countsort start\n");
	CountSort<<<block_num, thread_num_per_block>>>(darr, dhisto, size, max_val);
	printf("countsort end\n");
	
	int* histo = (int*)calloc(max_val, sizeof(int)); 
	hipMemcpy(histo, dhisto, sizeof(int)*max_val, hipMemcpyDeviceToHost);
	
	
	/*
	int cnt = 0;
	for(int i=0; i<max_val; i++) {
		cnt += histo[i];
	}
	printf("cnt: %d\n", cnt);
	*/
	
	int idx = 0;
	for(int i=0; i<max_val; i++) {
		for(int j=0; j<histo[i]; j++) {
			arr[idx++] = i;
		}
	}
	

	//cudaFree(dhisto);
	//cudaFree(darr);
	//free(histo);
}

__global__ void CountSort(int* darr, int* dhisto, int size, int max_val) {

	int thread_per_block = blockDim.x;
	int total_block = gridDim.x;
	int bid = blockIdx.x;
	int tid = threadIdx.x;
	uint64_t size_per_block, bstart, size_per_thread, start, end;


	// update histogram	in each block
	if(size % total_block != 0 && bid == total_block - 1) {
		size_per_block = size / total_block + size % total_block;
		bstart = bid * (size / total_block);
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && 
				tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}
	else {
		size_per_block = size / total_block;
		bstart = bid * size_per_block;	
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}

	for(uint64_t i=start; i<end; i++) {
		atomicAdd(&dhisto[(uint64_t)(darr[i] + bid * max_val)], 1);
	}
	__syncthreads();

	size_per_block = max_val;
	bstart = bid * size_per_block;
	size_per_thread = size_per_block / thread_per_block;
	start = bstart + tid * size_per_thread;
	end = start + size_per_thread;
	if(size_per_block % thread_per_block != 0 && tid == thread_per_block - 1) {
		end += size_per_block % thread_per_block;
	}
	if(bid != 0) {
		for(uint64_t i=start; i<end; i++) {
			atomicAdd(&dhisto[i%max_val], dhisto[i]);
		}
	}

	__syncthreads();

}

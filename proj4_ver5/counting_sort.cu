#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>

__global__ void CountSort(int*, int*, int, int);

__host__ void counting_sort(int* arr, int size, int max_val)
{
	int block_num = 1000;
	int thread_num_per_block = 1000;
	uint64_t histo_size = sizeof(int)*max_val;
	printf("size: %d\n", size);
	printf("max_val: %d\n", max_val);
	printf("block_num: %d\n", block_num);
	printf("thread_per_block: %d\n", thread_num_per_block);

	int* darr;
	hipMalloc(&darr, sizeof(int)*size);
	hipMemcpy(darr, arr, sizeof(int)*size, hipMemcpyHostToDevice); 

	int* dhisto;
	hipMalloc(&dhisto, histo_size);
	hipMemset(dhisto, 0, histo_size);

	printf("countsort start\n");
	CountSort<<<block_num, thread_num_per_block>>>(darr, dhisto, size, max_val);
	printf("countsort end\n");
	
	//int* histo = (int*)calloc(max_val, sizeof(int)); 
	//cudaMemcpy(histo, dhisto, sizeof(int)*max_val, cudaMemcpyDeviceToHost);
	hipMemcpy(arr, darr, sizeof(int)*size, hipMemcpyDeviceToHost);
	
	
	/*
	int cnt = 0;
	for(int i=0; i<max_val; i++) {
		cnt += histo[i];
	}
	printf("cnt: %d\n", cnt);
	*/
	
	/*
	int idx = 0;
	for(int i=0; i<max_val; i++) {
		for(int j=0; j<histo[i]; j++) {
			arr[idx++] = i;
		}
	}
	*/
	//cudaFree(dhisto);
	//cudaFree(darr);
	//free(histo);
	printf("return to main\n");
}

__global__ void CountSort(int* darr, int* dhisto, int size, int max_val) {

	uint64_t thread_per_block = blockDim.x;
	uint64_t total_block = gridDim.x;
	uint64_t bid = blockIdx.x;
	uint64_t tid = threadIdx.x;
	uint64_t size_per_block, bstart, size_per_thread, start, end;

	// update histogram	in each block
	if(size % total_block != 0 && bid == total_block - 1) {
		size_per_block = size / total_block + size % total_block;
		bstart = bid * (size / total_block);
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && 
				tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}
	else {
		size_per_block = size / total_block;
		bstart = bid * size_per_block;	
		size_per_thread = size_per_block / thread_per_block;
		start = bstart + tid * size_per_thread;
		end = start + size_per_thread;
		if(size_per_block % thread_per_block != 0 && tid == thread_per_block - 1) {
			end += size_per_block % thread_per_block;
		}
	}
	for(uint64_t i=start; i<end; i++) {
		atomicAdd(&dhisto[darr[i]], 1);
	}
	__syncthreads();


	if(bid == 0 && tid == 0) {
		/*
		for(int i=0; i<max_val; i++) {
			printf("histo[%d]: %d\n", i, dhisto[i]);
		}
		*/
		for(int i=1; i<max_val; i++) {
			dhisto[i] += dhisto[i-1];
		}
		/*
		for(int i=0; i<max_val; i++) {
			printf("histo[%d]: %d\n", i, dhisto[i]);
		}
		*/
	}
	__syncthreads();

	//printf("thread_per_block: %d\n", thread_per_block);
	size_per_block = max_val / total_block;		
	bstart = bid * size_per_block;
	size_per_thread = size_per_block / thread_per_block;
	start = bstart + tid * size_per_thread;
	end = start + size_per_thread;
	//printf("size_per_block: %d, thread_per_block: %d, bstart: %d, size_per_thread: %d\n", size_per_block, thread_per_block, bstart, size_per_thread);
	//printf("bid: %d, tid: %d => start: %d, end: %d\n", tid, bid, start, end);
	for(uint64_t i=start; i<end; i++) {
		darr[i] = i;
	}

	__syncthreads();
}
